#include "hip/hip_runtime.h"
#include "multiply_int32.hpp"

#include <cstddef>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

__global__ void matrixMultiplyKernel(const int32_t *A, const int32_t *B,
                                     int32_t *C, size_t N) {
  int32_t row = blockIdx.y * blockDim.y + threadIdx.y;
  int32_t col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < N && col < N) {
    int32_t sum = 0;
    for (int32_t k = 0; k < N; ++k) {
      sum += A[row * N + k] * B[k * N + col];
    }
    C[row * N + col] = sum;
  }
}

void matrixMultiplyCUDA(const int32_t *A, const int32_t *B, int32_t *C,
                        size_t N) {
  int32_t *d_A, *d_B, *d_C;
  size_t size = N * N * sizeof(int32_t);

  hipMalloc(&d_A, size);
  hipMalloc(&d_B, size);
  hipMalloc(&d_C, size);

  hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

  dim3 blockSize(16, 16);
  dim3 gridSize((N + blockSize.x - 1) / blockSize.x,
                (N + blockSize.y - 1) / blockSize.y);

  matrixMultiplyKernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);

  hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

// RNS
const int moduli[] = {2, 3, 5, 7, 11, 13, 17, 19};
const int num_moduli = sizeof(moduli) / sizeof(moduli[0]);
const int M = 2 * 3 * 5 * 7 * 11 * 13 * 17 * 19;

// Precomputed (M_i * inv_i mod M) (See misc/moduli.py)
const int term_i[] = {4849845, 3233230, 3879876, 8314020,
                      6172530, 3730650, 9129120, 9189180};

__global__ void convertToRNSKernel(const int32_t *A, int8_t *A_res,
                                   const int *d_moduli, size_t N) {
  size_t x = blockIdx.x * blockDim.x + threadIdx.x;
  size_t y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= N || y >= N)
    return;

  size_t idx = y * N + x;
  int32_t val = A[idx];

  for (int i = 0; i < num_moduli; ++i) {
    int m = d_moduli[i];
    int residue = val % m;
    if (residue < 0)
      residue += m;
    A_res[i * N * N + idx] = static_cast<int8_t>(residue);
  }
}

__global__ void matrixMulModKernel(const int8_t *A, const int8_t *B, int8_t *C,
                                   int m, size_t N) {
  size_t row = blockIdx.y * blockDim.y + threadIdx.y;
  size_t col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row >= N || col >= N)
    return;

  int8_t sum = 0;
  for (size_t k = 0; k < N; ++k) {
    int a = A[row * N + k];
    int b = B[k * N + col];

    sum = (sum + (a * b) % m) % m;
  }

  if (sum < 0)
    sum += m;

  C[row * N + col] = static_cast<int8_t>(sum);
}

__global__ void combineCRTKernel(const int8_t *C_res, int32_t *C,
                                 const int *d_term_i, size_t N) {
  size_t row = blockIdx.y * blockDim.y + threadIdx.y;
  size_t col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row >= N || col >= N)
    return;

  size_t idx = row * N + col;
  int32_t sum = 0;
  for (int i = 0; i < num_moduli; ++i) {
    int8_t c_i = C_res[i * N * N + idx];
    sum += c_i * d_term_i[i];
  }
  sum %= M;
  if (sum > M / 2)
    sum -= M;
  C[idx] = sum;
}

void rnsMatrixMultiply(const int32_t *h_A, const int32_t *h_B, int32_t *h_C,
                       size_t N) {
  int32_t *d_A, *d_B;
  int8_t *d_A_res, *d_B_res, *d_C_res;
  int32_t *d_C;
  int *d_moduli, *d_term_i;

  hipMalloc(&d_A, N * N * sizeof(int32_t));
  hipMalloc(&d_B, N * N * sizeof(int32_t));
  hipMemcpy(d_A, h_A, N * N * sizeof(int32_t), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, N * N * sizeof(int32_t), hipMemcpyHostToDevice);

  size_t rns_size = num_moduli * N * N * sizeof(int8_t);
  hipMalloc(&d_A_res, rns_size);
  hipMalloc(&d_B_res, rns_size);

  hipMalloc(&d_moduli, num_moduli * sizeof(int));
  hipMemcpy(d_moduli, moduli, num_moduli * sizeof(int),
             hipMemcpyHostToDevice);
  hipMalloc(&d_term_i, num_moduli * sizeof(int));
  hipMemcpy(d_term_i, term_i, num_moduli * sizeof(int),
             hipMemcpyHostToDevice);

  dim3 block(16, 16);
  dim3 grid((N + 15) / 16, (N + 15) / 16);
  convertToRNSKernel<<<grid, block>>>(d_A, d_A_res, d_moduli, N);
  convertToRNSKernel<<<grid, block>>>(d_B, d_B_res, d_moduli, N);
  hipDeviceSynchronize();

  hipMalloc(&d_C_res, rns_size);

  for (int i = 0; i < num_moduli; ++i) {
    matrixMulModKernel<<<grid, block>>>(d_A_res + i * N * N,
                                        d_B_res + i * N * N,
                                        d_C_res + i * N * N, moduli[i], N);
  }
  hipDeviceSynchronize();

  hipMalloc(&d_C, N * N * sizeof(int32_t));
  combineCRTKernel<<<grid, block>>>(d_C_res, d_C, d_term_i, N);
  hipDeviceSynchronize();

  hipMemcpy(h_C, d_C, N * N * sizeof(int32_t), hipMemcpyDeviceToHost);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_A_res);
  hipFree(d_B_res);
  hipFree(d_C_res);
  hipFree(d_C);
  hipFree(d_moduli);
  hipFree(d_term_i);
}
